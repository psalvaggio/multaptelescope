#include "hip/hip_runtime.h"
// File Description
// Author: Philip Salvaggio

#include "zernike_aberrations.h"
#include "zernike_cuda.h"

#include <cmath>
#include <iostream>

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#endif

using namespace cv;
using std::vector;

ZernikeAberrations::ZernikeAberrations() 
    : gpu_weights_(NULL), gpu_wfe_(NULL), gpu_wfe_size_(0) {
  #ifdef __HIPCC__
  hipMalloc(&gpu_weights_, 9 * sizeof(float));
  #endif
}

ZernikeAberrations::~ZernikeAberrations() {
  #ifdef __HIPCC__
  if (gpu_weights_) hipFree(gpu_weights_);
  if (gpu_wfe_) hipFree(gpu_wfe_);
  #endif
}

void ZernikeAberrations::aberrations(const vector<double>& weights,
                                     size_t output_size,
                                     Mat* output) {
  if (!output) return;

  const size_t kSize = output_size * output_size;
  
  #ifdef __HIPCC__

  const int kBlockSize = 1024;

  int num_blocks = (kSize % kBlockSize == 0)
      ? kSize / kBlockSize : kSize / kBlockSize + 1;

  float cpu_weights[9];
  for (int i = 0; i < 9; i++) {
    cpu_weights[i] = (i < weights.size()) ? weights[i] : 0;
  }

  if (gpu_weights_ == NULL) {
    hipMalloc(&gpu_weights_, 9 * sizeof(float));
  }
  hipMemcpy(gpu_weights_, cpu_weights, 9 * sizeof(float),
      hipMemcpyHostToDevice);

  if (output_size != gpu_wfe_size_ && gpu_wfe_) {
    hipFree(gpu_wfe_);
    gpu_wfe_ = NULL;
  }
  if (gpu_wfe_ == NULL) {
    hipMalloc(&gpu_wfe_, kSize * sizeof(float));
  }

  dim3 block, grid;
  block.x = kBlockSize; block.y = 1; block.z = 1;
  grid.x = num_blocks; grid.y = 1; grid.z = 1;
  zernike_kernel_4th<<<grid, block>>>(gpu_weights_, gpu_wfe_, output_size);

  output->create(output_size, output_size, CV_32FC1);
  hipMemcpy(output->data, gpu_wfe_, kSize * sizeof(float),
      hipMemcpyDeviceToHost);
  output->convertTo(*output, CV_64F);

  #else

  const double kCenter = 0.5 * (output_size - 1);

  output->create(output_size, output_size, CV_64F);
  *output = Scalar(0);

  double* output_data = reinterpret_cast<double*>(output->data);

  for (size_t i = 0; i < kSize; i++) {
    double x = (i % output_size) - kCenter;
    double y = (i / output_size) - kCenter;
    double rho = sqrt(x*x + y*y) / kCenter;

    if (rho > 1) {
      continue;
    }

    double theta = atan2(y, x);
    double cos_theta = cos(theta);
    double sin_theta = sin(theta);

    double wfe = 0;
    for (size_t j = 0; j < weights.size(); j++) {
      switch(j) {
        case 0:
          wfe += weights[0]; break;
        case 1:
          wfe += weights[1] * rho * cos_theta; break;
        case 2:
          wfe += weights[2] * rho * sin_theta; break;
        case 3:
          wfe += weights[3] * (rho*rho - 1); break;
        case 4:
          wfe += weights[4] * rho * rho * cos(2 * theta); break;
        case 5:
          wfe += weights[5] * rho * rho * sin(2 * theta); break;
        case 6:
          wfe += weights[6] * rho * (3 * rho * rho - 2) * cos_theta; break;
        case 7:
          wfe += weights[7] * rho * (3 * rho * rho - 2) * sin_theta; break;
        case 8:
          wfe += weights[8] * (1 - 6 * rho * rho + 6 * pow(rho, 4)); break;
        case 9:
          wfe += weights[9] * rho * rho * rho * cos(3 * theta); break;
        case 10:
          wfe += weights[10] * rho * rho * rho * sin(3 * theta); break;
        case 11:
          wfe += weights[11] * rho * rho * (4 * rho * rho - 3) * cos(2 * theta);
          break;
        case 12:
          wfe += weights[12] * rho * rho * (4 * rho * rho - 3) * sin(2 * theta);
          break;
        case 13:
          wfe += weights[13] * rho * (3 - 12 * rho*rho + 10 * pow(rho, 4)) *
              cos_theta;
          break;
        case 14:
          wfe += weights[14] * rho * (3 - 12 * rho*rho + 10 * pow(rho, 4)) *
              sin_theta;
          break;
        case 15:
          wfe += weights[15] * (-1 + 12 * rho*rho - 30 * pow(rho, 4) + 20 *
              pow(rho, 6));
          break;
        case 16:
          wfe += weights[16] * pow(rho, 4) * cos(4 * theta); break;
        case 17:
          wfe += weights[17] * pow(rho, 4) * sin(4 * theta); break;
        case 18:
          wfe += weights[18] * pow(rho, 3) * (5 * rho*rho - 4) * cos(3 * theta);
          break;
        case 19:
          wfe += weights[19] * pow(rho, 3) * (5 * rho*rho - 4) * sin(3 * theta);
          break;
        case 20:
          wfe += weights[20] * rho * rho * (6 - 20 * rho*rho + 15 *
              pow(rho, 4)) * cos(2 * theta);
          break;
        case 21:
          wfe += weights[21] * rho * rho * (6 - 20 * rho*rho + 15 *
              pow(rho, 4)) * sin(2 * theta);
          break;
        case 22:
          wfe += weights[22] * rho * (-4 + 30 * rho*rho - 60 * pow(rho, 4) +
              35 * pow(rho, 6)) * cos(theta);
          break;
        case 23:
          wfe += weights[23] * rho * (-4 + 30 * rho*rho - 60 * pow(rho, 4) +
              35 * pow(rho, 6)) * sin(theta);
          break;
        case 24:
          wfe += weights[24] * (1 - 20 * rho*rho + 90 * pow(rho, 4) - 140 *
              pow(rho, 6) + 70 * pow(rho, 8));
          break;
      }
    }
    output_data[i] = wfe;
  }
  
  #endif
}

#ifdef __HIPCC__
__global__
void zernike_kernel_4th(float* weights,
                        float* output,
			int size) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  float center = 0.5 * (size - 1);

  float x = (index % size) - center;
  float y = (index / size) - center;
  float rho = sqrt(x*x + y*y) / center;
  float rho2 = rho * rho;

  float theta = atan2(y, x);
  float cos_theta = cos(theta);
  float sin_theta = sin(theta);

  float wfe =
      weights[0] +
      weights[1] * rho * cos_theta +
      weights[2] * rho * sin_theta +
      weights[3] * (rho2 - 1) +
      weights[4] * rho2 * cos(2 * theta) +
      weights[5] * rho2 * sin(2 * theta) +
      weights[6] * rho * (3 * rho2 - 2) * cos_theta +
      weights[7] * rho * (3 * rho2 - 2) * sin_theta +
      weights[8] * (1 - 6 * rho2 + 6 * rho2 * rho2);

  if (rho <= 1 && index < size * size) {
    output[index] = wfe;
  } else {
    output[index] = 0;
  }
}

#endif
